#include "hip/hip_runtime.h"
#include "volume.h"

int main(int argc, char *argv[]) {
  FILE *fp = NULL;
  tSTL input;
  uint threads, blocks, sum_threads;
  float cpu_result, *gpu_result, tmp_result, *reduce;
  tMesh cpu_mesh, gpu_mesh;

  // timing stuff
  struct timeval t1, t2;
  hipEvent_t start, stop;
  float dt_cpu, dt_gpu;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // parse arguments
  if (argc != 3) {
    fprintf(stderr, "Usage: volume <n> <file>\n");
    fprintf(stderr, "\tn\tnumber of threads per block (32,64,128,256,512,1024)\n");
    fprintf(stderr, "\tfile\tpath to an STL file\n");
    return -1;
  }
  // validate threads per block
  threads = atoi(argv[1]);
  switch (threads) {
    case 32:
    case 64:
    case 128:
    case 256:
    case 512:
    case 1024:
      break;
    default:
      fprintf(stderr, "Wrong number of threads per block!\n");
      return -1;
  }
  // open input file
  fp = fopen(argv[2], "rb");
  if (fp == NULL) {
    fprintf(stderr, "Input file could not be opened!\n");
    return -1;
  }

  // read file header
  fseek(fp, sizeof(char) * 80, SEEK_SET);
  fread(&cpu_mesh.num, sizeof(uint32_t), 1, fp);

  // allocate CPU mesh
  cpu_mesh.a = (float4 *) malloc(sizeof(float4) * cpu_mesh.num);
  cpu_mesh.b = (float4 *) malloc(sizeof(float4) * cpu_mesh.num);
  cpu_mesh.c = (float4 *) malloc(sizeof(float4) * cpu_mesh.num);

  // read the triangles from file
  for (int i=0; i<cpu_mesh.num; i++) {
    fread(&input, sizeof(tSTL), 1, fp);
    cpu_mesh.a[i].x = input.points[0];
    cpu_mesh.a[i].y = input.points[1];
    cpu_mesh.a[i].z = input.points[2];
    cpu_mesh.b[i].x = input.points[3];
    cpu_mesh.b[i].y = input.points[4];
    cpu_mesh.b[i].z = input.points[5];
    cpu_mesh.c[i].x = input.points[6];
    cpu_mesh.c[i].y = input.points[7];
    cpu_mesh.c[i].z = input.points[8];
  }

  fclose(fp);

  // calculate reference solution on CPU
  gettimeofday(&t1, 0);
  volume_calculate_cpu(cpu_mesh, &cpu_result);
  gettimeofday(&t2, 0);
  dt_cpu = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000.0;

  // set parameters for kernel
  blocks = ceil(((float)cpu_mesh.num) / ((float)threads));
  gpu_mesh.num = threads * blocks;

  sum_threads = ceil(((float)blocks) / ((float)threads)) * threads;
  sum_threads = pow(2, ceil(log(sum_threads) / log(2)));

  // allocate
  hipMalloc(&gpu_mesh.a, sizeof(float4) * gpu_mesh.num);
  hipMalloc(&gpu_mesh.b, sizeof(float4) * gpu_mesh.num);
  hipMalloc(&gpu_mesh.c, sizeof(float4) * gpu_mesh.num);
  // copy
  hipMemcpy(gpu_mesh.a, cpu_mesh.a, sizeof(float4) * cpu_mesh.num, hipMemcpyHostToDevice);
  hipMemcpy(gpu_mesh.b, cpu_mesh.b, sizeof(float4) * cpu_mesh.num, hipMemcpyHostToDevice);
  hipMemcpy(gpu_mesh.c, cpu_mesh.c, sizeof(float4) * cpu_mesh.num, hipMemcpyHostToDevice);
  // set the padding
  hipMemset(&gpu_mesh.a[cpu_mesh.num], 0, sizeof(float4) * (gpu_mesh.num - cpu_mesh.num));
  hipMemset(&gpu_mesh.b[cpu_mesh.num], 0, sizeof(float4) * (gpu_mesh.num - cpu_mesh.num));
  hipMemset(&gpu_mesh.c[cpu_mesh.num], 0, sizeof(float4) * (gpu_mesh.num - cpu_mesh.num));

  // allocate memory for the results
  hipMalloc(&gpu_result, sizeof(float) * sum_threads);
  hipMemset(&gpu_result[blocks], 0, sizeof(float) * (sum_threads - blocks));
  hipMalloc(&reduce, sizeof(float) * gpu_mesh.num);

  // invoke kernel
  hipEventRecord(start, 0);
  switch (threads) {
    case 1024:
      volume_calculate_gpu<1024> <<<blocks,threads,sizeof(float4)*threads>>>(gpu_mesh, gpu_result);
      hipDeviceSynchronize();
      sum_calculate<1024> <<<1,threads,sizeof(float)*sum_threads>>>(gpu_result, sum_threads);
      break;
    case 512:
      volume_calculate_gpu<512> <<<blocks,threads,sizeof(float4)*threads>>>(gpu_mesh, gpu_result);
      hipDeviceSynchronize();
      sum_calculate<512> <<<1,threads,sizeof(float)*sum_threads>>>(gpu_result, sum_threads);
      break;
    case 256:
      volume_calculate_gpu<256> <<<blocks,threads,sizeof(float4)*threads>>>(gpu_mesh, gpu_result);
      hipDeviceSynchronize();
      sum_calculate<256> <<<1,threads,sizeof(float)*sum_threads>>>(gpu_result, sum_threads);
      break;
    case 128:
      volume_calculate_gpu<128> <<<blocks,threads,sizeof(float4)*threads>>>(gpu_mesh, gpu_result);
      hipDeviceSynchronize();
      sum_calculate<128> <<<1,threads,sizeof(float)*sum_threads>>>(gpu_result, sum_threads);
      break;
    case 64:
      volume_calculate_gpu<64> <<<blocks,threads,sizeof(float4)*threads>>>(gpu_mesh, gpu_result);
      hipDeviceSynchronize();
      sum_calculate<64> <<<1,threads,sizeof(float)*sum_threads>>>(gpu_result, sum_threads);
      break;
    case 32:
      volume_calculate_gpu<32> <<<blocks,threads,sizeof(float4)*threads>>>(gpu_mesh, gpu_result);
      hipDeviceSynchronize();
      sum_calculate<32> <<<1,threads,sizeof(float)*sum_threads>>>(gpu_result, sum_threads);
      break;
  }
  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(start);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&dt_gpu, start, stop);

  // copy back and sum
  hipMemcpy(&tmp_result, gpu_result, sizeof(float), hipMemcpyDeviceToHost);

  // print results
  printf("Number of triangles %d, padded in GPU to %d\n", cpu_mesh.num, gpu_mesh.num);
  printf("Volume calculated by CPU: %0.3f in %fms\n", abs(cpu_result), dt_cpu);
  printf("Volume calculated by GPU: %0.3f in %fms\n", abs(tmp_result), dt_gpu);

  // clean up
  free(cpu_mesh.a);
  free(cpu_mesh.b);
  free(cpu_mesh.c);
  hipFree(gpu_mesh.a);
  hipFree(gpu_mesh.b);
  hipFree(gpu_mesh.c);
  hipFree(gpu_result);
  hipFree(reduce);
}